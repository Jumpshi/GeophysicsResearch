/*
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/*
* Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/* Example showing the use of CUFFT for fast 1D-convolution using FFT. */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>
#include "cu_stft.h"
// Complex data type
////////////////////////////////////////////////////////////////////////////////
void my_cufft(Complex *h_signal, int SIGNAL_SIZE)
{
	// Allocate device memory for signal
	int mem_size = sizeof(Complex)*SIGNAL_SIZE;
	Complex *d_signal;
	hipMalloc((void **)&d_signal, mem_size);
	// Copy host memory to device
	hipMemcpy(d_signal, h_signal, mem_size,
		hipMemcpyHostToDevice);

	hipMemcpy(h_signal, d_signal, mem_size,
		hipMemcpyDeviceToHost);
	// CUFFT plan simple API
	hipfftHandle plan;
	hipfftPlan1d(&plan, SIGNAL_SIZE, HIPFFT_C2C, 1);
	// CUFFT plan advanced API
	hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD);
	hipMemcpy(h_signal, d_signal, mem_size,
		hipMemcpyDeviceToHost);
	//Destroy CUFFT context
	hipfftDestroy(plan);
	hipFree(d_signal);
}

Complex *my_stft(Complex *h_signal, int s_size, int w_lag, int s_len)
{
	// Allocate device memory for signal
	//int mem_size = sizeof(Complex)*SIGNAL_SIZE;
	int x_size = (int)((s_size - s_len) / w_lag);
	int y_size = s_len;
	int stft_size = sizeof(Complex)*x_size*y_size;
	int mem_size = sizeof(Complex)*s_size;
	Complex *d_signal;
	hipMalloc((void **)&d_signal, mem_size);
	// Copy host memory to device
	hipMemcpy(d_signal, h_signal, mem_size,
		hipMemcpyHostToDevice);
	Complex *d_stft;
	hipMalloc((void **)&d_stft, stft_size);
	//hipMemcpy(h_signal, d_signal, mem_size,
		//hipMemcpyDeviceToHost);
	// CUFFT plan simple API
	hipfftHandle plan;
	hipfftPlan1d(&plan, s_len, HIPFFT_C2C, 1);
	// CUFFT plan advanced API
	for (int i = 0; i < x_size; i++)
	{
		hipfftExecC2C(plan, (hipfftComplex *)d_signal+i, (hipfftComplex *)d_stft+i*y_size, HIPFFT_FORWARD);
	}
	Complex *h_stft;
	h_stft = (Complex *)malloc(sizeof(Complex)*stft_size);
	hipMemcpy(h_stft, d_stft, stft_size,
		hipMemcpyDeviceToHost);
	//Destroy CUFFT context
	hipfftDestroy(plan);
	hipFree(d_signal);
	hipFree(d_stft);
	return h_stft;
}